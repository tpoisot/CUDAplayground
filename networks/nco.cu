/*
 * Null model based on cell link strength
 * ranged between 0 and 1
 * 
 * CUDA version
 * 
 * compile with nvcc nco.cu -o nco -lcuda
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

__global__ void nullmodel(float *M, int *out, hiprandState *states)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(threadIdx.x, 0, 0, &states[tid]);
	float tar = (float)hiprand_uniform(&states[tid]);
	out[tid] = (tar < M[tid])? 1 : 0;
}

int main(int argc, char **argv)
{
	srand ( time(NULL) );
	clock_t start, stop;

	// PRNG
	hiprandState *devStates;

	// Network shape
	int nrow = 100;
	int ncol = 100;
	double connec = 0.6;
	int net_size = nrow * ncol; 

	// Memory allocation on host
	const size_t nbytes_ref= net_size * sizeof(float);
	const size_t nbytes_out= net_size * sizeof(int);
	float *h_ref = (float*)malloc(nbytes_ref);
	int *h_out = (int*)malloc(nbytes_out);

	// create a random betwork -------
	for(int row = 0; row < nrow; ++row)
	{
		for(int col = 0; col < ncol; ++col)
		{
			if((double)(rand() / (float)RAND_MAX) < connec)
			{
				h_ref[col + row*nrow] = (float)(rand() / (float)RAND_MAX);
			}
		}
	}
	// rand() / (float)RAND_MAX;	

	// Memory allocation on device
	float *d_ref;
	int *d_out;
	hipMalloc((void**)&d_ref, nbytes_ref);
	hipMalloc((void**)&d_out, nbytes_out);

	// Memory transfer from host to device
	hipMemcpy(d_ref, h_ref, nbytes_ref, hipMemcpyHostToDevice);
	hipMemcpy(d_out, h_out, nbytes_out, hipMemcpyHostToDevice);
	hipMalloc( (void **)&devStates, net_size * sizeof(hiprandState) );

	// Record initial time and start doing the null model
	start = clock();

	for(int repl = 0; repl < 1000; ++repl)
	{
		nullmodel<<<ncol,nrow>>>(d_ref, d_out, devStates);
		// Memory transfer from the device to the host
		hipMemcpy(h_out, d_out, nbytes_ref, hipMemcpyDeviceToHost);
	}
	
	stop = clock();
	printf("1000 null network generated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	// Free memory space
	free(h_ref);
	free(h_out);
	hipFree(d_ref);
	hipFree(d_out);

	return EXIT_SUCCESS;
}