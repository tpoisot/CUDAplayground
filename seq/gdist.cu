// nvcc gdist.cu -o gdist -lcuda -use_fast_math compiler

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
  
#define NS 50
#define NBASES 100
#define seqpos(s_,b_)  ((b_) + (s_) * NBASES)

__global__ void pwdist(char *se, float *dist)
{
	int tid = blockIdx.x;
	int M = blockDim.x - 1;
	int ii = (M * (M - 1)) / 2 - tid - 1;
	int K = (sqrtf(8 * ii + 1) - 1) / 2;
	
	int FSidx = M + K;
	int SCidx = tid - M*(M+1)/2 + (K+1)*(K+2)/2; 
	
	int Dissim = 0;
	for(int base = 0; base < NBASES; ++base)
	{
		if(!(se[seqpos(FSidx,base)] == se[seqpos(SCidx,base)]))
		{
			++Dissim;
		}
	}
	
	dist[blockIdx.x] = Dissim;
	
    //int tid = threadIdx.x + blockDim.x * blockIdx.x; // blockDim.x := 1000
    
}

int main (int argc, char *argv[])
{	
	// generate a bunch of test sequences
	int ds = NS * NBASES;	
	const unsigned int bytes = ds * sizeof(char);
	char *se = (char*)malloc(bytes);
	
	// create cyclic sequences
	for(int i = 0; i < ds; ++i)
	{
		se[i] = 'a';
		//if (i % 3 == 4){se[i] = 'c';}
		//if (i % 7 == 4){se[i] = 't';}
		//if (i % 19 == 8){se[i] = '-';}
		//if (i % 13 == 1){se[i] = 'g';}
	}
	
	// print sequences to screen just to check
	for(int i = 0; i < NBASES; i++)
	{
		for(int j = 0; j < NS; j++)
		{
			printf("%c",se[seqpos(j,i)]);
		}
		printf("\n");
	}
	
	// number of distances to compute
	const unsigned int ndist = (int)(NS*(NS-1)/2);
	// memory allocation for the distances
	float *h_dist = (float*)malloc(ndist*sizeof(float));
	
	// allocate memory on the devices
	char *d_se;
	float *d_dist;
	hipMalloc((void**)&d_se, bytes);
	hipMalloc((void**)&d_dist, ndist * sizeof(float));
	
	/*
	 * HERE BE THE MAIN PROGRAM
	 */
	
	// copy the sequences from the host to the device
	hipMemcpy(d_se, se, bytes, hipMemcpyHostToDevice);
	
	// call CUDA function pwdist
	pwdist<<<ndist,1>>>(d_se, d_dist);
	
	// copy the distances from the device to the host
	hipMemcpy(h_dist, d_dist, ndist * sizeof(float), hipMemcpyDeviceToHost);
	
	/*
	 * END OF THE MAIN PROGRAM
	 */
	
	// Output of the results in a text form
	for(int i = 0; i < ndist; ++i)
	{
		printf("%f \t",h_dist[i]);
	}
	
	// free the vectors
	hipFree(d_se);
	hipFree(d_dist);
	free(se);
	free(h_dist);
	
	return EXIT_SUCCESS;
}